#include "hip/hip_runtime.h"
﻿//#include "cuda_def.cuh"
//#include "cuda_proc.h"
#include "mergeClusteringGB.h"
//////////////////////////////////
//function
//////////////////////////////////
template <class T>
void convertFromString(T &value, string &s)
{
    std::stringstream ss(s);
    ss >> value;
}

void findMeredCluster(const vector<vector<float>> &distMatrix, const float &threshold, const int &sampNum, 
                      vector<int> &mergedIndexList)
    {
        vector<set<int>> lowThresIndex;
        /*************first schem, merging speed is low***************/
        // for(int i = 0; i < distMatrix.size(); i++)
        // {
        //     set <int> lowIndexList;
        //     for(int j = 0; j < distMatrix[i].size(); j++)
        //     {   
        //         int merIndex = (int) j/sampNum;
        //         if(distMatrix[i][j]<threshold+1) lowIndexList.insert(merIndex);
        //     }
        //     lowThresIndex.push_back(lowIndexList);
        // }
        /*************first schem, merging speed is low***************/
        
        for(int i = 0; i < distMatrix.size(); i++)
        {
            set <int> lowIndexList;
            for(int j = 0; j < distMatrix[i].size(); j+=sampNum)
            {
                vector<float> splitDistList(distMatrix[i].begin()+j, distMatrix[i].begin()+j+sampNum);
                sort(splitDistList.begin(), splitDistList.end());
                int Num = sampNum > 5? 5 : sampNum;
                float sumDist = accumulate(std::begin(splitDistList), splitDistList.begin()+Num, 0.0);
                float fthres = sumDist / Num;
                // float sumDist = accumulate(std::begin(splitDistList), std::end(splitDistList), 0.0);
                // float meanDist = sumDist / sampNum;
                // float maxDist = *max_element(std::begin(splitDistList), std::end(splitDistList));
                // float minDist = *min_element(std::begin(splitDistList), std::end(splitDistList));
                // float fthres = (meanDist + maxDist) / 2;
                // float fthres = (minDist + maxDist) / 2;
                // float fthres = maxDist / 4;
                if(fthres<threshold+1) 
                {   
                    int merIndex = (int) j/sampNum;
                    lowIndexList.insert(merIndex);
                }
            }
            lowThresIndex.push_back(lowIndexList);
        }

        set<int> succList(lowThresIndex[0].begin(), lowThresIndex[0].end());
        for(int i = 1; i < lowThresIndex.size(); i++)
        {
            set_intersection(succList.begin(),succList.end(),
            lowThresIndex[i].begin(),lowThresIndex[i].end(),inserter(succList,succList.begin()));
        }

        // mergedIndexList = succList;
        mergedIndexList.assign(succList.begin(), succList.end());
        // for(int i = 0; i < succList.size(); i++) mergedIndexList.push_back(succList[i]);
    }

int majorElemCandidate(const vector<int> A)
{
    int maj;
    int count(0);
    for(int i = 0; i < A.size(); ++i){
        if(count == 0){
            maj = A[i];
            count++;
        } else {
            maj == A[i]? count++ : count--;
        }
    }
    return maj; 
}
    

void findMinSize(vector<vector<int> > vec, int &minSize)
{
    minSize = vec[0].size();
    for(int i = 0; i < vec.size(); i++)
    {   
        int len = vec[i].size();
        if(minSize > len) minSize = len;
    }
}

void randomSample(vector<int> &vec, const int &sampleNum, vector<int> &resList)
{   
    // srand ( unsigned ( time(0) ) );
    random_shuffle(vec.begin(), vec.end());
    for(int i = 0; i < sampleNum; i++)
    {
        resList.push_back(vec[i]);
    }
}

void oneT2D(vector<float> &oneDList, vector<vector<float>> &twoDList)
{
    for(int i = 0; i < twoDList.size(); i++)
    {
        for(int j = 0; j < twoDList[i].size(); j++)
        {
            oneDList.push_back(twoDList[i][j]);
        }
    }
}

void fromClusterGetEle(vector<int> &oneDList, vector<vector<int>> &twoDList)
{
    for(int i = 0; i < twoDList.size(); i++)
    {
        for(int j = 0; j < twoDList[i].size(); j++)
        {
            oneDList.push_back(twoDList[i][j]);
        }
    }
}

void DeleteNoteOff(vector<vector<int> > &eventStore)
{
   eventStore.erase(std::remove_if(eventStore.begin(), eventStore.end(), 
                    [](const std::vector<int>& v) {return v.size() > 1 && v[v.size()-1] == -1;}), 
                    eventStore.end());
}


vector<vector<float>> transpose(vector<vector<float>>& A) 
{
    int leny=A[0].size();
    int lenx=A.size();
    vector<vector<float>> v(leny, vector<float>(lenx, 0));
    if(A.empty()) return vector<vector<float>>();
    for(int i=0;i<lenx;i++)
        for(int j=0;j<leny;j++)
        {
            v[j][i] = A[i][j]; 
        }
    return v;
}

void readClusterFile(vector<int> &goodClusterList, vector<int> &badClusterList, const string &clusterFile)
{
    ifstream cluster;
    cluster.open(clusterFile, ios::in);
    if (!cluster.is_open())
    {
        cout << "Cluster merge failed! cluster file does not exist!" << endl;
        exit(-1);
    }
    string FileLine;
    int t = 0;
    while(getline(cluster, FileLine))
    {   
        istringstream is(FileLine);
        int index;
        while (!is.eof()) 
        {
			is >> index;
            if(t == 0)
            {
                goodClusterList.push_back(index);
            }
            else
            {
                badClusterList.push_back(index);
            }
			
		}
        t += 1;
    }
    cluster.close();
    goodClusterList.pop_back();
    badClusterList.pop_back();
}

void readGoodClusterFile(vector<vector<int> > &goodClusterList, const string &goodClusterFile)
{
    ifstream cluster;
    cluster.open(goodClusterFile, ios::in);
    if (!cluster.is_open())
    {
        cout << "Good cluster merge failed! Good cluster file does not exist!" << endl;
        exit(-1);
    }
    string FileLine;
    while(getline(cluster, FileLine))
    {   
        vector<int> singleCluster;
        istringstream is(FileLine);
        int index;
        while (!is.eof()) 
        {   
            is >> index;
			singleCluster.push_back(index);
		}
        singleCluster.pop_back();
        goodClusterList.push_back(singleCluster);
    }
    cluster.close();
}

void TestDataGeneration(vector<vector<float>> &siga, vector<vector<float>> &sigb, int siga_n,
                        int sigb_n, bool aeb)
{
    siga.resize(siga_n);
    sigb.resize(sigb_n);
  
    omp_set_num_threads(CPUTHREADS);
#pragma omp parallel for
    for (int i = 0; i < siga_n; i++)
    {
        int siga_length = rand() % 200 + 700; 
        siga[i].resize(siga_length);
        for (int j = 0; j < siga_length; j++)
        {
            siga[i][j] = rand() % 400 + 400; 
        }
    }
    if (siga_n == sigb_n && aeb)
    {
        for (int i = 0; i < sigb_n; i++)
        {
            sigb[i] = siga[i];
        }
        return;
    }
#pragma omp parallel for
    for (int i = 0; i < sigb_n; i++)
    {
        int sigb_length = rand() % 200 + 700; 
        sigb[i].resize(sigb_length);
        for (int j = 0; j < sigb_length; j++)
        {
            sigb[i][j] = rand() % 400 + 400; 
        }
    }
}

float gpuCalculatemnDynamicTimeWarping_1024(const vector<vector<float>> &siga,
                                            const vector<vector<float>> &sigb,
                                            vector<vector<float>> &gpudistResult)
{
    // return 0;
    int siga_n = siga.size();
    int sigb_n = sigb.size();
    gpudistResult.resize(siga_n);
    for (int i = 0; i < siga_n; i++)
    {
        gpudistResult[i].resize(sigb_n);
    }
    // int siga_length = 0;
    float *d_distResult = NULL;
    float *d_allColData = NULL;
    float *d_allRowData = NULL;
    unsigned int *d_allRowLength;
    unsigned int *d_allColLength;
    // vector<float *> rowDataList(sigb_n);
    hipMalloc((void **)&d_distResult, siga_n * sigb_n * sizeof(float));
    CUERR
    hipMalloc((void **)&d_allColData, siga_n * 1024 * sizeof(float));
    CUERR
    hipMalloc((void **)&d_allRowData, sigb_n * 1024 * sizeof(float));
    CUERR

    vector<unsigned int> h_allRowLength(sigb_n);
    for (int i = 0; i < sigb_n; i++)
    {
        h_allRowLength[i] = sigb[i].size();
        hipMemcpy(&d_allRowData[1024 * i], &sigb[i][0], h_allRowLength[i] * sizeof(float),
                   hipMemcpyHostToDevice);
        CUERR
    }
    hipMalloc((void **)&d_allRowLength, sigb_n * sizeof(unsigned int));
    CUERR
    hipMemcpy(d_allRowLength, &h_allRowLength[0], sigb_n * sizeof(unsigned int),
               hipMemcpyHostToDevice);
    CUERR

    vector<unsigned int> h_allColLength(siga_n);
    for (int i = 0; i < siga_n; i++)
    {
        h_allColLength[i] = min(int(siga[i].size()), 1024);
        hipMemcpy(&d_allColData[1024 * i], &siga[i][0], h_allColLength[i] * sizeof(float),
                   hipMemcpyHostToDevice);
        CUERR
    }
    hipMalloc((void **)&d_allColLength, siga_n * sizeof(unsigned int));
    CUERR
    hipMemcpy(d_allColLength, &h_allColLength[0], siga_n * sizeof(unsigned int),
               hipMemcpyHostToDevice);
    CUERR

    float timesum = 0;
    dim3 threadsPerBlock(1024);
    dim3 blocksPerGrid(sigb_n, siga_n);
    cuDTW_1024<<<blocksPerGrid, threadsPerBlock>>>(d_allColData, d_allColLength, d_allRowData,
                                                   d_allRowLength, d_distResult);
    CUERR

    for (int i = 0; i < siga_n; i++)
    {
        hipMemcpy(&gpudistResult[i][0], &d_distResult[sigb_n * i], sigb_n * sizeof(float),
                   hipMemcpyDeviceToHost);
        CUERR
    }

    hipFree(d_allColData);
    CUERR
    hipFree(d_distResult);
    CUERR
    hipFree(d_allRowData);
    CUERR
    hipFree(d_allRowLength);
    CUERR

    return timesum;
}

float cpuCalculatemnDynamicTimeWarping(const vector<vector<float>> &siga,
                                       const vector<vector<float>> &sigb,
                                       vector<vector<float>> &cpudistResult)
{

    int siga_n = siga.size();
    int sigb_n = sigb.size();
    cpudistResult.resize(siga_n);
    for (int i = 0; i < siga_n; i++)
    {
        cpudistResult[i].resize(sigb_n);
    }
    omp_set_num_threads(CPUTHREADS);
#pragma omp parallel for
    for (int i = 0; i < siga_n; i++)
    {
        for (int j = 0; j < sigb_n; j++)
        {
            cpudistResult[i][j] = cpuDynamicTimeWarping(siga[i], sigb[j]);
        }
    }

    float timesum = 0;
    // printf("cpu Average time use of DTW= %f sec\n", timesum / sigb_n);
    return timesum;
}

float cpuDynamicTimeWarping(const std::vector<float> &seq1, const std::vector<float> &seq2)
{
    vector<vector<float>> score(seq1.size());

    for (int i = 0; i < seq1.size(); i++)
    {
        score[i].resize(seq2.size());
    }

    for (int i = 0; i < seq1.size(); i++)
    {
        for (int j = 0; j < seq2.size(); j++)
        {
            score[i][j] = std::fabs(seq1[i] - seq2[j]);
        }
    }

    for (int i = 1; i < seq1.size(); i++)
    {
        score[i][0] += score[i - 1][0];
    }

    for (int j = 1; j < seq2.size(); j++)
    {
        score[0][j] += score[0][j - 1];
    }

    for (int i = 1; i < seq1.size(); i++)
    {
        for (int j = 1; j < seq2.size(); j++)
        {
            score[i][j] +=
                std::min(std::min(score[i - 1][j], score[i][j - 1]), score[i - 1][j - 1]);
        }
    }

    float diff = score[seq1.size() - 1][seq2.size() - 1];

    return diff;
}
vector<float> GetSingleSignalData(int &SignalId, const string &argv_1, const string &sigRootName)
{
    // cout << SignalId << endl;
    ifstream SignalFile;
    string TempId = std::to_string(SignalId);
    // string TempId2 = std::to_string(SignalScale);
    string TempString = argv_1 + "/" + sigRootName + "_" + TempId + ".txt";
    // cout << TempString << endl;
    const char *FileName = TempString.data();
    SignalFile.open(FileName, ios::in);
    if (!SignalFile.is_open())
    {
        cout << "Signal file open error!" << endl;
        cout << argv_1 << endl;
        cout << SignalId << endl;
    }
    string FileLine;
    vector<float> SignalData;
    while (getline(SignalFile, FileLine))
    {
        float SignalValue;
        convertFromString(SignalValue, FileLine);
        SignalData.push_back(SignalValue);
    }
    ZScoreNormalize(SignalData);
    SignalFile.close();
    return SignalData;
}

void getSigsOfList(vector<vector<float> > &res, vector<int> &idxList, const string &sigDirPath, const string &sigRootName)
{
    for(int i = 0; i < idxList.size(); i++)
    {
        vector<float> sig = GetSingleSignalData(idxList[i], sigDirPath, sigRootName);
        res.push_back(sig);
    }
}

// vector<string> GetargvList()
// {
//     ifstream argvfile;
//     // const char* filename = "ReadyToSort.txt".data();
//     argvfile.open("argv_file.txt", ios::in);
//     if (!argvfile.is_open())
//     {
//         cout << "argv_file.txt open error!" << endl;
//     }
//     string FileLine;
//     vector<string> argvList;
//     while (getline(argvfile, FileLine))
//     {
//         argvList.push_back(FileLine);
//     }
//     return argvList;
// }

// vector<vector<float>> GetCenterSignalList(const int &SignalScale)
// {
//     vector<vector<float>> SignalList;
//     for (int i = 0; i < SignalScale; i++)
//     {
//         vector<float> TempSignal = GetSingleSignalData(i, SignalScale);
//         SignalList.push_back(TempSignal);
//     }
//     return SignalList;
// }

void refineOneEleCluster(const vector<vector<float>> &sigb, float &threshold, float &devideIndex,
                        vector<vector<int>> &gpuclusterResult, int maxLocalLength)
{
    vector<vector<float>> siga, remainSigb(sigb);
    int sigb_n = sigb.size();
    int remainSigb_n = remainSigb.size();
    int siga_n = 0;
    vector<int> label(remainSigb_n), aindex(remainSigb_n), bindex(remainSigb_n),
        used(remainSigb_n, 0);

    {
        for (int i = 0; i < remainSigb_n; i++)
        {
            label[i] = i;
            bindex[i] = i;
        }
    }

    for (int loop = 0; loop < 10; loop++)
    {
        // printf("loop=%d-----------------------\n", loop);
        if (loop > 0)
        {
            remainSigb.clear();
            for (int i = 0; i < sigb_n; i++)
            {
                if (used[i] == 0)
                {
                    remainSigb.push_back(sigb[i]);
                    bindex[remainSigb.size() - 1] = i;
                }
            }
        }
        remainSigb_n = remainSigb.size();
        // printf("sigb_length=%d\n", remainSigb_n);
        if (remainSigb.size() == 0)
        {
            break;
        }
        {
            int randnum;
            siga.clear();
            for (int i = 0; i < remainSigb_n; i++)
            {
                randnum = rand() % 1000;
                // printf("%d ", randnum);
                
                if (randnum < 100000.0 / remainSigb_n)
                {
                    siga.push_back(remainSigb[i]);
                    aindex[siga.size() - 1] = bindex[i];
                    // printf("picked i=%d\n", i);
                }
            }
            // printf("siga_length=%d\n", siga.size());
        }

        siga_n = siga.size();
        vector<vector<float>> gpudistResult;
        if (maxLocalLength == 1024)
        {
            gpuCalculatemnDynamicTimeWarping_1024(siga, remainSigb, gpudistResult);
        }
        else if (maxLocalLength == 2048)
        {
            gpuCalculatemnDynamicTimeWarping_2048(siga, remainSigb, gpudistResult);
        }
        else
        {
            printf("wrong max local length!\n");
            exit(-2);
        }
      
        float maxVal = *max_element(gpudistResult[0].begin(), gpudistResult[0].end());
        float minVal = *min_element(gpudistResult[0].begin(), gpudistResult[0].end());
        float threshold = (maxVal + minVal) / devideIndex;
            // printf("maxDist,minDist,thredhold=%f %f %f\n", maxVal, minVal, THRESHOLD);
        

        for (int i = 0; i < siga_n; i++)
        {
            if (used[aindex[i]] == 0)
            {
                for (int j = 0; j < remainSigb_n; j++)
                {
                    if (gpudistResult[i][j] < threshold)
                    {
                        label[bindex[j]] = label[aindex[i]];
                        used[bindex[j]] = 1;
                    }
                }
            }
        }
    }
    for (int i = 0; i < sigb_n; i++)
    {
        // if (i % CLUSTERNUM == 0) {
        //     printf("\n");
        // }
        // printf("%d ", label[i]);
        bool notFound = true;
        for (int j = 0; j < gpuclusterResult.size(); j++)
        {
            if (label[i] == gpuclusterResult[j][0])
            {
                if (label[i] != i)
                {
                    gpuclusterResult[j].push_back(i);
                }
                notFound = false;
                break;
            }
        }
        if (notFound)
        {
            vector<int> thisCluster;
            thisCluster.push_back(label[i]);
            if (label[i] != i)
            {
                thisCluster.push_back(i);
            }
            gpuclusterResult.push_back(thisCluster);
        }
    }
    // printf("cluster numbers=%d\n", gpuclusterResult.size());

    // for (int i = 0; i < gpuclusterResult.size(); i++) {
    //     for (int j = 0; j < gpuclusterResult[i].size(); j++) {
    //         printf("%d ", gpuclusterResult[i][j]);
    //     }
        // ENDL;
    // }
    // printf("\n");
}

void ZScoreNormalize(std::vector<float> &signals, float *avg, float *stdev)
{
    // CLOCKSTART
    float sum = std::accumulate(signals.begin(), signals.end(), 0.0);
    float mean = sum / signals.size();

    float acc = 0.0;
    for (size_t i = signals.size(); i--;)
    {
        signals[i] = signals[i] - mean;
        acc += signals[i] * signals[i];
    }

    float deviation = std::sqrt(acc / signals.size());

    for (size_t i = signals.size(); i--;)
    {
        signals[i] /= deviation;
    }

    if (avg)
    {
        *avg = mean;
    }
    if (stdev)
    {
        *stdev = deviation;
    }
    // printf("ZScoreNorm ");
    // CLOCKSTOP
}

float gpuCalculatemnDynamicTimeWarping_2048(const vector<vector<float>> &siga,
                                            const vector<vector<float>> &sigb,
                                            vector<vector<float>> &gpudistResult)
{
    // return 0;
    int siga_n = siga.size();
    int sigb_n = sigb.size();
    gpudistResult.resize(siga_n);
    for (int i = 0; i < siga_n; i++)
    {
        gpudistResult[i].resize(sigb_n);
    }
    // int siga_length = 0;
    float *d_distResult = NULL;
    float *d_allColData = NULL;
    float *d_allRowData = NULL;
    unsigned int *d_allRowLength;
    unsigned int *d_allColLength;
    // vector<float *> rowDataList(sigb_n);
    hipMalloc((void **)&d_distResult, siga_n * sigb_n * sizeof(float));
    CUERR
    hipMalloc((void **)&d_allColData, siga_n * 2048 * sizeof(float));
    CUERR
    hipMalloc((void **)&d_allRowData, sigb_n * 2048 * sizeof(float));
    CUERR

    
    vector<unsigned int> h_allRowLength(sigb_n);
    for (int i = 0; i < sigb_n; i++)
    {
        h_allRowLength[i] = min(int(sigb[i].size()), 2048);
        hipMemcpy(&d_allRowData[2048 * i], &sigb[i][0], h_allRowLength[i] * sizeof(float),
                   hipMemcpyHostToDevice);
        CUERR
    }
    hipMalloc((void **)&d_allRowLength, sigb_n * sizeof(unsigned int));
    CUERR
    hipMemcpy(d_allRowLength, &h_allRowLength[0], sigb_n * sizeof(unsigned int),
               hipMemcpyHostToDevice);
    CUERR

   
    vector<unsigned int> h_allColLength(siga_n);
    for (int i = 0; i < siga_n; i++)
    {
        h_allColLength[i] = min(int(siga[i].size()), 2048);
        hipMemcpy(&d_allColData[2048 * i], &siga[i][0], h_allColLength[i] * sizeof(float),
                   hipMemcpyHostToDevice);
        CUERR
    }
    hipMalloc((void **)&d_allColLength, siga_n * sizeof(unsigned int));
    CUERR
    hipMemcpy(d_allColLength, &h_allColLength[0], siga_n * sizeof(unsigned int),
               hipMemcpyHostToDevice);
    CUERR

    float timesum = 0;
    dim3 threadsPerBlock(1024);
    dim3 blocksPerGrid(sigb_n, siga_n); 
    cuDTW_2048<<<blocksPerGrid, threadsPerBlock>>>(d_allColData, d_allColLength, d_allRowData,
                                                   d_allRowLength, d_distResult);
    CUERR

    for (int i = 0; i < siga_n; i++)
    {
        hipMemcpy(&gpudistResult[i][0], &d_distResult[sigb_n * i], sigb_n * sizeof(float),
                   hipMemcpyDeviceToHost);
        CUERR
    }

    hipFree(d_allColData);
    CUERR
    hipFree(d_distResult);
    CUERR
    hipFree(d_allRowData);
    CUERR
    hipFree(d_allRowLength);
    CUERR

    return timesum;
}

__global__ void cuDTW_1024(float *g_allColData, unsigned int *g_allColLength, float *g_allRowData,
                           unsigned int *g_allRowLength, float *g_odata)
{

    unsigned int rowLength = g_allRowLength[blockIdx.x]; 
    unsigned int colLength = g_allColLength[blockIdx.y];
    float myNum = 0, myColNum;
    __shared__ unsigned int s_turn;
    __shared__ float preNum[1024], prepreNum[1024], rowData[1024];
    rowData[threadIdx.x] =
        g_allRowData[blockIdx.x * 1024 + threadIdx.x]; 
    if (threadIdx.x == 0)
    {
        s_turn = 0;
    }
    __syncthreads();
    if (threadIdx.x < colLength)
    {
        myColNum = g_allColData[blockIdx.y * 1024 + threadIdx.x];
        prepreNum[threadIdx.x] = preNum[threadIdx.x] = 0;
        int col;
        while (s_turn < colLength + rowLength)
        {
            col = s_turn - threadIdx.x;
            if (col >= 0 && col < rowLength)
            {
                if (threadIdx.x == 0)
                {
                    myNum = preNum[threadIdx.x] + fabs(myColNum - rowData[col]);
                }
                else
                {
                    if (col == 0)
                    {
                        myNum = preNum[threadIdx.x - 1] + fabs(myColNum - rowData[col]);
                    }
                    else
                    {
                        myNum = min(min(prepreNum[threadIdx.x - 1], preNum[threadIdx.x - 1]),
                                    preNum[threadIdx.x]) +
                                fabs(myColNum - rowData[col]);
                    }
                }
            }
            __syncthreads();
            prepreNum[threadIdx.x] = preNum[threadIdx.x];
            preNum[threadIdx.x] = myNum;
            if (threadIdx.x == 0)
            {
                // printf("--\nI am first thread of %d block,myIdx=%d,turn=%d,col=%d\n",
                // blockIdx.x,
                //      threadIdx.x, s_turn, col);
                s_turn++;
            }
            __syncthreads();
        }
    }
    if (threadIdx.x == colLength - 1)
    {
        // if (myNum < 50000)
        //    printf("my blockIdx=%d,my result=%f\n", blockIdx.x,
        //    preNum[threadIdx.x]);
        g_odata[blockIdx.y * gridDim.x + blockIdx.x] = myNum;
    }
}

__global__ void cuDTW_2048(float *g_allColData, unsigned int *g_allColLength, float *g_allRowData,
                           unsigned int *g_allRowLength, float *g_odata)
{

    unsigned int rowLength = g_allRowLength[blockIdx.x]; // block的x是sigb，block的y是siga
    unsigned int colLength = g_allColLength[blockIdx.y];
    float myNum1 = 0, myNum2 = 0, myColNum1, myColNum2;
    __shared__ unsigned int s_turn;
    __shared__ float preNum1[1024], preNum2[1024], prepreNum2[1024], rowData[2048];
    
    rowData[threadIdx.x] = g_allRowData[blockIdx.x * 2048 + threadIdx.x];
    __syncthreads();
    rowData[threadIdx.x + 1024] = g_allRowData[blockIdx.x * 2048 + threadIdx.x + 1024];
    if (threadIdx.x == 0)
    {
        s_turn = 0;
    }
    __syncthreads();
    if (threadIdx.x < (colLength - 1) / 2 + 1)
    {
        
        myColNum1 = g_allColData[blockIdx.y * 2048 + (threadIdx.x) * 2];
        myColNum2 = g_allColData[blockIdx.y * 2048 + (threadIdx.x) * 2 + 1];
        prepreNum2[threadIdx.x] = preNum2[threadIdx.x] = preNum1[threadIdx.x] = 0;
        int col;
        while (s_turn < (colLength - 1) / 2 + rowLength)
        {                               
            col = s_turn - threadIdx.x; 
            if (col >= 0 && col < rowLength)
            {
                if (threadIdx.x == 0)
                {
                    myNum1 = preNum1[0] + fabs(myColNum1 - rowData[col]);

                    if (col == 0)
                    { 
                        myNum2 = myNum1 + fabs(myColNum2 - rowData[col]);
                    }
                    else
                    {
                        myNum2 = min(min(myNum1, preNum1[0]), preNum2[0]) +
                                 fabs(myColNum2 - rowData[col]);
                    }
                }
                else
                {
                    if (col == 0)
                    {
                        myNum1 = preNum2[threadIdx.x - 1] + fabs(myColNum1 - rowData[col]);
                        myNum2 = myNum1 + fabs(myColNum2 - rowData[col]);
                    }
                    else
                    {
                        myNum1 = min(min(prepreNum2[threadIdx.x - 1], preNum2[threadIdx.x - 1]),
                                     preNum1[threadIdx.x]) +
                                 fabs(myColNum1 - rowData[col]);
                        myNum2 = min(min(myNum1, preNum1[threadIdx.x]), preNum2[threadIdx.x]) +
                                 fabs(myColNum2 - rowData[col]);
                    }
                }
            }
            __syncthreads();
            prepreNum2[threadIdx.x] = preNum2[threadIdx.x];
            preNum2[threadIdx.x] = myNum2;
            preNum1[threadIdx.x] = myNum1;
            if (threadIdx.x == 0)
            {
                // printf("--\nI am first thread of %d block,myIdx=%d,turn=%d,col=%d\n",
                // blockIdx.x,
                //      threadIdx.x, s_turn, col);
                s_turn++;
            }
            __syncthreads();
        }
    }
    if (threadIdx.x == (colLength - 1) / 2)
    {
        // if (myNum1 < 50000)
        //    printf("my blockIdx=%d,my result=%f\n", blockIdx.x,
        //    preNum[threadIdx.x]);
        if (colLength % 2 == 0)
            g_odata[blockIdx.y * gridDim.x + blockIdx.x] = myNum2;
        else
            g_odata[blockIdx.y * gridDim.x + blockIdx.x] = myNum1;
    }
}
 

// vector<vector<float>> GetAlignSignalList(vector<int> &SignalIdList, const int &SignalScale)
// {
//     vector<vector<float>> CenterSignalList;
//     for (int i = 2; i < SignalIdList.size(); i++)
//     {
//         vector<float> TempSignal = GetSingleSignalData(SignalIdList[i], SignalScale);
//         CenterSignalList.push_back(TempSignal);
//     }
//     return CenterSignalList;
// }