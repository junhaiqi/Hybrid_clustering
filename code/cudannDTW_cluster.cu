#include "hip/hip_runtime.h"
﻿//#include "cuda_def.cuh"
//#include "cuda_proc.h"
#include "cuda_def.cuh"
#include <algorithm>
#include <fstream>
#include <iostream>
#include <numeric>
#include <sstream>
#include <string>
#include <vector>
//#include "cuda_kernels.h"
#include <omp.h>
#include <vector>
#define DBL_MAX 1.7976931348623158e+308 // max value
#define PI 3.1415926535898
#define MAX_NUM 2147483647
#define MAXLENGTH 1023
#define CPUTHREADS 36
int CLUSTERNUM = 10;
float THRESHOLD = 90;
#define ENDL printf("\n");

using namespace std;
//////////////////////////////////
//function
//////////////////////////////////

vector<vector<float>> GetAlignSignalList_1(vector<int> &SignalIdList, const string &argv_1);

__global__ void cuDTW_1024(float *g_allColData, unsigned int *g_allColLength, float *g_allRowData,
                           unsigned int *g_allRowLength, float *g_odata);
__global__ void cuDTW_2048(float *g_allColData, unsigned int *g_allColLength, float *g_allRowData,
                           unsigned int *g_allRowLength, float *g_odata);
void TestDataGeneration(vector<vector<float>> &siga, vector<vector<float>> &sigb, int siga_n = 50,
                        int sigb_n = 50, bool aeb = false);
float gpuCalculatemnDynamicTimeWarping_1024(const vector<vector<float>> &siga,
                                            const vector<vector<float>> &sigb,
                                            vector<vector<float>> &gpudistResult);
float gpuCalculatemnDynamicTimeWarping_2048(const vector<vector<float>> &siga,
                                            const vector<vector<float>> &sigb,
                                            vector<vector<float>> &gpudistResult);
float cpuCalculatemnDynamicTimeWarping(const vector<vector<float>> &siga,
                                       const vector<vector<float>> &sigb,
                                       vector<vector<float>> &cpudistResult);
float cpuDynamicTimeWarping(const std::vector<float> &seq1, const std::vector<float> &seq2);
void gpuCluster(const vector<vector<float>> &sigb, float threshold,
                vector<vector<int>> &gpuclusterResult, int maxLocalLength);

vector<int> GetAlignList();

vector<vector<double>> GetAlignSignalList(vector<int> &SignalIdList, const string &argv_1);

vector<string> GetargvList();

vector<float> GetSingleSignalData(int &SignalId, const string &argv_1);

vector<vector<float>> GetCenterSignalList(const int &SignalScale);
void ZScoreNormalize(std::vector<float> &signals, float *avg = NULL, float *stdev = NULL);
template <class T>
void convertFromString(T &value, string &s)
{
    std::stringstream ss(s);
    ss >> value;
}

// using namespace std;

int main(int argc, char **argv)
{
    vector<vector<float>> siga;
    vector<vector<float>> sigb;

    int siga_n = 200;
    int sigb_n = 10000;
    if (argc >= 3)
    {
        siga_n = atoi(argv[1]);
        sigb_n = atoi(argv[2]);
        if (argc == 4)
        {
            THRESHOLD = atoi(argv[3]);
        }
    }

    float gputimeuse, cputimeuse;

    // printf("reading data...\n");

    vector<int> Alignlist = GetAlignList();

    vector<string> argvList = GetargvList();

    sigb = GetAlignSignalList_1(Alignlist, argvList[0]);

    // printf("TestDataGeneration Finished!... \n");
    vector<vector<int>> realgpuclusterResult;
    vector<vector<int>> gpuclusterResult;
    {

        int mod = 2048;
        // printf("choose a mod: 1024 or 2048 ? ");
        // cin >> mod;
        // printf("\nstart cuda cluster... \n");
        CLOCKSTART
        gpuCluster(sigb, Alignlist[1], gpuclusterResult, mod);
        // printf("cuda cluster ");
        CLOCKSTOP
    }

    // cout << gpuclusterResult[0][1] << endl;
    // cout << Alignlist.size() << endl;

    vector<int> templist;
    for (int p = 0; p < gpuclusterResult.size(); p++)
    {
        vector<int> templist;
        for (int q = 0; q < gpuclusterResult[p].size(); q++)
        {
            // cout << i;
            // cout << gpuclusterResult[p][q] << endl;
            int temp = gpuclusterResult[p][q] + 2;
            // cout << gpuclusterResult[p][q] << endl;
            // cout << Alignlist[temp] << endl;
            templist.push_back(Alignlist[temp]);
            // cout << realgpuclusterResult[p][q] << endl;
            // realgpuclusterResult[p].push_back(Alignlist[temp]);
            // cout << "--------------------";
            // cout <<  realgpuclusterResult[i][j] << endl;
            // cout << 1 << endl;
        }
        realgpuclusterResult.push_back(templist);
    }

    vector<vector<float>> gpudistResult;
    vector<vector<float>> cpudistResult;

    ofstream disfile("gpuClusterResult.txt", ios::out);
    string temp;
    for (int i = 0; i < realgpuclusterResult.size(); i++)
    {
        for (int j = 0; j < realgpuclusterResult[i].size(); j++)
        {
            temp = std::to_string(realgpuclusterResult[i][j]);
            disfile << temp;
            disfile << " ";
        }
        disfile << endl;
    }
    // cout << gpudtwresult.size() << endl;
    disfile.close();

    // exit(0);

    return 0;
}

//////////////////////////////////
// function
//////////////////////////////////

vector<vector<float>> GetAlignSignalList_1(vector<int> &SignalIdList, const string &argv_1)
{

    vector<vector<float>> CenterSignalList;
    for (int i = 2; i < SignalIdList.size(); i++)
    {
        vector<float> TempSignal = GetSingleSignalData(SignalIdList[i], argv_1);
        CenterSignalList.push_back(TempSignal);
    }
    return CenterSignalList;
}

vector<int> GetAlignList()
{
    ifstream Alignfile;
    // const char* filename = "ReadyToSort.txt".data();
    Alignfile.open("ReadyToSortfile_1.txt", ios::in);
    if (!Alignfile.is_open())
    {
        cout << "Align file open error!" << endl;
    }
    string FileLine;
    vector<int> AlignList;
    while (getline(Alignfile, FileLine))
    {
        int SignalId;
        convertFromString(SignalId, FileLine);
        AlignList.push_back(SignalId);
    }
    return AlignList;
}

void TestDataGeneration(vector<vector<float>> &siga, vector<vector<float>> &sigb, int siga_n,
                        int sigb_n, bool aeb)
{
    siga.resize(siga_n);
    sigb.resize(sigb_n);
  
    omp_set_num_threads(CPUTHREADS);
#pragma omp parallel for
    for (int i = 0; i < siga_n; i++)
    {
        int siga_length = rand() % 200 + 700; 
        siga[i].resize(siga_length);
        for (int j = 0; j < siga_length; j++)
        {
            siga[i][j] = rand() % 400 + 400; 
        }
    }
    if (siga_n == sigb_n && aeb)
    {
        for (int i = 0; i < sigb_n; i++)
        {
            sigb[i] = siga[i];
        }
        return;
    }
#pragma omp parallel for
    for (int i = 0; i < sigb_n; i++)
    {
        int sigb_length = rand() % 200 + 700; 
        sigb[i].resize(sigb_length);
        for (int j = 0; j < sigb_length; j++)
        {
            sigb[i][j] = rand() % 400 + 400; 
        }
    }
}

float gpuCalculatemnDynamicTimeWarping_1024(const vector<vector<float>> &siga,
                                            const vector<vector<float>> &sigb,
                                            vector<vector<float>> &gpudistResult)
{
    // return 0;
    int siga_n = siga.size();
    int sigb_n = sigb.size();
    gpudistResult.resize(siga_n);
    for (int i = 0; i < siga_n; i++)
    {
        gpudistResult[i].resize(sigb_n);
    }
    // int siga_length = 0;
    float *d_distResult = NULL;
    float *d_allColData = NULL;
    float *d_allRowData = NULL;
    unsigned int *d_allRowLength;
    unsigned int *d_allColLength;
    // vector<float *> rowDataList(sigb_n);
    hipMalloc((void **)&d_distResult, siga_n * sigb_n * sizeof(float));
    CUERR
    hipMalloc((void **)&d_allColData, siga_n * 1024 * sizeof(float));
    CUERR
    hipMalloc((void **)&d_allRowData, sigb_n * 1024 * sizeof(float));
    CUERR

    vector<unsigned int> h_allRowLength(sigb_n);
    for (int i = 0; i < sigb_n; i++)
    {
        h_allRowLength[i] = sigb[i].size();
        hipMemcpy(&d_allRowData[1024 * i], &sigb[i][0], h_allRowLength[i] * sizeof(float),
                   hipMemcpyHostToDevice);
        CUERR
    }
    hipMalloc((void **)&d_allRowLength, sigb_n * sizeof(unsigned int));
    CUERR
    hipMemcpy(d_allRowLength, &h_allRowLength[0], sigb_n * sizeof(unsigned int),
               hipMemcpyHostToDevice);
    CUERR

    vector<unsigned int> h_allColLength(siga_n);
    for (int i = 0; i < siga_n; i++)
    {
        h_allColLength[i] = min(int(siga[i].size()), 1024);
        hipMemcpy(&d_allColData[1024 * i], &siga[i][0], h_allColLength[i] * sizeof(float),
                   hipMemcpyHostToDevice);
        CUERR
    }
    hipMalloc((void **)&d_allColLength, siga_n * sizeof(unsigned int));
    CUERR
    hipMemcpy(d_allColLength, &h_allColLength[0], siga_n * sizeof(unsigned int),
               hipMemcpyHostToDevice);
    CUERR

    float timesum = 0;
    dim3 threadsPerBlock(1024);
    dim3 blocksPerGrid(sigb_n, siga_n);
    cuDTW_1024<<<blocksPerGrid, threadsPerBlock>>>(d_allColData, d_allColLength, d_allRowData,
                                                   d_allRowLength, d_distResult);
    CUERR

    for (int i = 0; i < siga_n; i++)
    {
        hipMemcpy(&gpudistResult[i][0], &d_distResult[sigb_n * i], sigb_n * sizeof(float),
                   hipMemcpyDeviceToHost);
        CUERR
    }

    hipFree(d_allColData);
    CUERR
    hipFree(d_distResult);
    CUERR
    hipFree(d_allRowData);
    CUERR
    hipFree(d_allRowLength);
    CUERR

    return timesum;
}

float cpuCalculatemnDynamicTimeWarping(const vector<vector<float>> &siga,
                                       const vector<vector<float>> &sigb,
                                       vector<vector<float>> &cpudistResult)
{

    int siga_n = siga.size();
    int sigb_n = sigb.size();
    cpudistResult.resize(siga_n);
    for (int i = 0; i < siga_n; i++)
    {
        cpudistResult[i].resize(sigb_n);
    }
    omp_set_num_threads(CPUTHREADS);
#pragma omp parallel for
    for (int i = 0; i < siga_n; i++)
    {
        for (int j = 0; j < sigb_n; j++)
        {
            cpudistResult[i][j] = cpuDynamicTimeWarping(siga[i], sigb[j]);
        }
    }

    float timesum = 0;
    // printf("cpu Average time use of DTW= %f sec\n", timesum / sigb_n);
    return timesum;
}

float cpuDynamicTimeWarping(const std::vector<float> &seq1, const std::vector<float> &seq2)
{
    vector<vector<float>> score(seq1.size());

    for (int i = 0; i < seq1.size(); i++)
    {
        score[i].resize(seq2.size());
    }

    for (int i = 0; i < seq1.size(); i++)
    {
        for (int j = 0; j < seq2.size(); j++)
        {
            score[i][j] = std::fabs(seq1[i] - seq2[j]);
        }
    }

    for (int i = 1; i < seq1.size(); i++)
    {
        score[i][0] += score[i - 1][0];
    }

    for (int j = 1; j < seq2.size(); j++)
    {
        score[0][j] += score[0][j - 1];
    }

    for (int i = 1; i < seq1.size(); i++)
    {
        for (int j = 1; j < seq2.size(); j++)
        {
            score[i][j] +=
                std::min(std::min(score[i - 1][j], score[i][j - 1]), score[i - 1][j - 1]);
        }
    }

    float diff = score[seq1.size() - 1][seq2.size() - 1];

    return diff;
}
vector<float> GetSingleSignalData(int &SignalId, const string &argv_1)
{
    // cout << SignalId << endl;
    ifstream SignalFile;
    string TempId = std::to_string(SignalId);
    // string TempId2 = std::to_string(SignalScale);
    string TempString = argv_1 + "/" + "signal_" + TempId + ".txt";
    const char *FileName = TempString.data();
    SignalFile.open(FileName, ios::in);
    if (!SignalFile.is_open())
    {
        cout << "Signal file open error!" << endl;
        cout << argv_1 << endl;
        cout << SignalId << endl;
    }
    string FileLine;
    vector<float> SignalData;
    while (getline(SignalFile, FileLine))
    {
        float SignalValue;
        convertFromString(SignalValue, FileLine);
        SignalData.push_back(SignalValue);
    }
    ZScoreNormalize(SignalData);
    SignalFile.close();
    return SignalData;
}

vector<string> GetargvList()
{
    ifstream argvfile;
    // const char* filename = "ReadyToSort.txt".data();
    argvfile.open("argv_file.txt", ios::in);
    if (!argvfile.is_open())
    {
        cout << "argv_file.txt open error!" << endl;
    }
    string FileLine;
    vector<string> argvList;
    while (getline(argvfile, FileLine))
    {
        argvList.push_back(FileLine);
    }
    return argvList;
}

// vector<vector<float>> GetCenterSignalList(const int &SignalScale)
// {
//     vector<vector<float>> SignalList;
//     for (int i = 0; i < SignalScale; i++)
//     {
//         vector<float> TempSignal = GetSingleSignalData(i, SignalScale);
//         SignalList.push_back(TempSignal);
//     }
//     return SignalList;
// }

void gpuCluster(const vector<vector<float>> &sigb, float threshold,
                vector<vector<int>> &gpuclusterResult, int maxLocalLength)
{
    vector<vector<float>> siga, remainSigb(sigb);
    int sigb_n = sigb.size();
    int remainSigb_n = remainSigb.size();
    int siga_n, usedCount = 0;
    vector<int> label(remainSigb_n), aindex(remainSigb_n), bindex(remainSigb_n),
        used(remainSigb_n, 0);

    {
        for (int i = 0; i < remainSigb_n; i++)
        {
            label[i] = i;
            bindex[i] = i;
        }
    }

    for (int loop = 0; loop < 10; loop++)
    {
        // printf("loop=%d-----------------------\n", loop);
        if (loop > 0)
        {
            remainSigb.clear();
            for (int i = 0; i < sigb_n; i++)
            {
                if (used[i] == 0)
                {
                    remainSigb.push_back(sigb[i]);
                    bindex[remainSigb.size() - 1] = i;
                }
            }
        }
        remainSigb_n = remainSigb.size();
        // printf("sigb_length=%d\n", remainSigb_n);
        if (remainSigb.size() == 0)
        {
            break;
        }
        {
            //通过随机的方式来初始化这一次要用的数据
            int randnum;
            siga.clear();
            for (int i = 0; i < remainSigb_n; i++)
            {
                randnum = rand() % 1000;
                // printf("%d ", randnum);
                
                if (randnum < 100000.0 / remainSigb_n)
                {
                    siga.push_back(remainSigb[i]);
                    aindex[siga.size() - 1] = bindex[i];
                    // printf("picked i=%d\n", i);
                }
            }
            // printf("siga_length=%d\n", siga.size());
        }

        siga_n = siga.size();
        vector<vector<float>> gpudistResult;
        if (maxLocalLength == 1024)
        {
            gpuCalculatemnDynamicTimeWarping_1024(siga, remainSigb, gpudistResult);
        }
        else if (maxLocalLength == 2048)
        {
            gpuCalculatemnDynamicTimeWarping_2048(siga, remainSigb, gpudistResult);
        }
        else
        {
            printf("wrong max local length!\n");
            exit(-2);
        }
        if (loop == 0)
        {

            float maxVal = *max_element(gpudistResult[0].begin(), gpudistResult[0].end());
            float minVal = *min_element(gpudistResult[0].begin(), gpudistResult[0].end());
            THRESHOLD = (maxVal + minVal) / 2;
            // printf("maxDist,minDist,thredhold=%f %f %f\n", maxVal, minVal, THRESHOLD);
        }

        for (int i = 0; i < siga_n; i++)
        {
            if (used[aindex[i]] == 0)
            {
                for (int j = 0; j < remainSigb_n; j++)
                {
                    if (gpudistResult[i][j] < THRESHOLD)
                    {
                        label[bindex[j]] = label[aindex[i]];
                        used[bindex[j]] = 1;
                    }
                }
            }
        }
    }
    for (int i = 0; i < sigb_n; i++)
    {
        // if (i % CLUSTERNUM == 0) {
        //     printf("\n");
        // }
        // printf("%d ", label[i]);
        bool notFound = true;
        for (int j = 0; j < gpuclusterResult.size(); j++)
        {
            if (label[i] == gpuclusterResult[j][0])
            {
                if (label[i] != i)
                {
                    gpuclusterResult[j].push_back(i);
                }
                notFound = false;
                break;
            }
        }
        if (notFound)
        {
            vector<int> thisCluster;
            thisCluster.push_back(label[i]);
            if (label[i] != i)
            {
                thisCluster.push_back(i);
            }
            gpuclusterResult.push_back(thisCluster);
        }
    }
    // printf("cluster numbers=%d\n", gpuclusterResult.size());

    // // for (int i = 0; i < gpuclusterResult.size(); i++) {
    // //     for (int j = 0; j < gpuclusterResult[i].size(); j++) {
    // //         printf("%d ", gpuclusterResult[i][j]);
    // //     }
    //     ENDL;
    // // }

    // printf("\n");
}
void ZScoreNormalize(std::vector<float> &signals, float *avg, float *stdev)
{
    // CLOCKSTART
    float sum = std::accumulate(signals.begin(), signals.end(), 0.0);
    float mean = sum / signals.size();

    float acc = 0.0;
    for (size_t i = signals.size(); i--;)
    {
        signals[i] = signals[i] - mean;
        acc += signals[i] * signals[i];
    }

    float deviation = std::sqrt(acc / signals.size());

    for (size_t i = signals.size(); i--;)
    {
        signals[i] /= deviation;
    }

    if (avg)
    {
        *avg = mean;
    }
    if (stdev)
    {
        *stdev = deviation;
    }
    // printf("ZScoreNorm ");
    // CLOCKSTOP
}

float gpuCalculatemnDynamicTimeWarping_2048(const vector<vector<float>> &siga,
                                            const vector<vector<float>> &sigb,
                                            vector<vector<float>> &gpudistResult)
{
    // return 0;
    int siga_n = siga.size();
    int sigb_n = sigb.size();
    gpudistResult.resize(siga_n);
    for (int i = 0; i < siga_n; i++)
    {
        gpudistResult[i].resize(sigb_n);
    }
    // int siga_length = 0;
    float *d_distResult = NULL;
    float *d_allColData = NULL;
    float *d_allRowData = NULL;
    unsigned int *d_allRowLength;
    unsigned int *d_allColLength;
    // vector<float *> rowDataList(sigb_n);
    hipMalloc((void **)&d_distResult, siga_n * sigb_n * sizeof(float));
    CUERR
    hipMalloc((void **)&d_allColData, siga_n * 2048 * sizeof(float));
    CUERR
    hipMalloc((void **)&d_allRowData, sigb_n * 2048 * sizeof(float));
    CUERR

    
    vector<unsigned int> h_allRowLength(sigb_n);
    for (int i = 0; i < sigb_n; i++)
    {
        h_allRowLength[i] = min(int(sigb[i].size()), 2048);
        hipMemcpy(&d_allRowData[2048 * i], &sigb[i][0], h_allRowLength[i] * sizeof(float),
                   hipMemcpyHostToDevice);
        CUERR
    }
    hipMalloc((void **)&d_allRowLength, sigb_n * sizeof(unsigned int));
    CUERR
    hipMemcpy(d_allRowLength, &h_allRowLength[0], sigb_n * sizeof(unsigned int),
               hipMemcpyHostToDevice);
    CUERR

   
    vector<unsigned int> h_allColLength(siga_n);
    for (int i = 0; i < siga_n; i++)
    {
        h_allColLength[i] = min(int(siga[i].size()), 2048);
        hipMemcpy(&d_allColData[2048 * i], &siga[i][0], h_allColLength[i] * sizeof(float),
                   hipMemcpyHostToDevice);
        CUERR
    }
    hipMalloc((void **)&d_allColLength, siga_n * sizeof(unsigned int));
    CUERR
    hipMemcpy(d_allColLength, &h_allColLength[0], siga_n * sizeof(unsigned int),
               hipMemcpyHostToDevice);
    CUERR

    float timesum = 0;
    dim3 threadsPerBlock(1024);
    dim3 blocksPerGrid(sigb_n, siga_n); 
    cuDTW_2048<<<blocksPerGrid, threadsPerBlock>>>(d_allColData, d_allColLength, d_allRowData,
                                                   d_allRowLength, d_distResult);
    CUERR

    for (int i = 0; i < siga_n; i++)
    {
        hipMemcpy(&gpudistResult[i][0], &d_distResult[sigb_n * i], sigb_n * sizeof(float),
                   hipMemcpyDeviceToHost);
        CUERR
    }

    hipFree(d_allColData);
    CUERR
    hipFree(d_distResult);
    CUERR
    hipFree(d_allRowData);
    CUERR
    hipFree(d_allRowLength);
    CUERR

    return timesum;
}

__global__ void cuDTW_1024(float *g_allColData, unsigned int *g_allColLength, float *g_allRowData,
                           unsigned int *g_allRowLength, float *g_odata)
{

    unsigned int rowLength = g_allRowLength[blockIdx.x]; 
    unsigned int colLength = g_allColLength[blockIdx.y];
    float myNum = 0, myColNum;
    __shared__ unsigned int s_turn;
    __shared__ float preNum[1024], prepreNum[1024], rowData[1024];
    rowData[threadIdx.x] =
        g_allRowData[blockIdx.x * 1024 + threadIdx.x]; 
    if (threadIdx.x == 0)
    {
        s_turn = 0;
    }
    __syncthreads();
    if (threadIdx.x < colLength)
    {
        myColNum = g_allColData[blockIdx.y * 1024 + threadIdx.x];
        prepreNum[threadIdx.x] = preNum[threadIdx.x] = 0;
        int col;
        while (s_turn < colLength + rowLength)
        {
            col = s_turn - threadIdx.x;
            if (col >= 0 && col < rowLength)
            {
                if (threadIdx.x == 0)
                {
                    myNum = preNum[threadIdx.x] + fabs(myColNum - rowData[col]);
                }
                else
                {
                    if (col == 0)
                    {
                        myNum = preNum[threadIdx.x - 1] + fabs(myColNum - rowData[col]);
                    }
                    else
                    {
                        myNum = min(min(prepreNum[threadIdx.x - 1], preNum[threadIdx.x - 1]),
                                    preNum[threadIdx.x]) +
                                fabs(myColNum - rowData[col]);
                    }
                }
            }
            __syncthreads();
            prepreNum[threadIdx.x] = preNum[threadIdx.x];
            preNum[threadIdx.x] = myNum;
            if (threadIdx.x == 0)
            {
                // printf("--\nI am first thread of %d block,myIdx=%d,turn=%d,col=%d\n",
                // blockIdx.x,
                //      threadIdx.x, s_turn, col);
                s_turn++;
            }
            __syncthreads();
        }
    }
    if (threadIdx.x == colLength - 1)
    {
        // if (myNum < 50000)
        //    printf("my blockIdx=%d,my result=%f\n", blockIdx.x,
        //    preNum[threadIdx.x]);
        g_odata[blockIdx.y * gridDim.x + blockIdx.x] = myNum;
    }
}

__global__ void cuDTW_2048(float *g_allColData, unsigned int *g_allColLength, float *g_allRowData,
                           unsigned int *g_allRowLength, float *g_odata)
{

    unsigned int rowLength = g_allRowLength[blockIdx.x]; // block的x是sigb，block的y是siga
    unsigned int colLength = g_allColLength[blockIdx.y];
    float myNum1 = 0, myNum2 = 0, myColNum1, myColNum2;
    __shared__ unsigned int s_turn;
    __shared__ float preNum1[1024], preNum2[1024], prepreNum2[1024], rowData[2048];
    
    rowData[threadIdx.x] = g_allRowData[blockIdx.x * 2048 + threadIdx.x];
    __syncthreads();
    rowData[threadIdx.x + 1024] = g_allRowData[blockIdx.x * 2048 + threadIdx.x + 1024];
    if (threadIdx.x == 0)
    {
        s_turn = 0;
    }
    __syncthreads();
    if (threadIdx.x < (colLength - 1) / 2 + 1)
    {
        
        myColNum1 = g_allColData[blockIdx.y * 2048 + (threadIdx.x) * 2];
        myColNum2 = g_allColData[blockIdx.y * 2048 + (threadIdx.x) * 2 + 1];
        prepreNum2[threadIdx.x] = preNum2[threadIdx.x] = preNum1[threadIdx.x] = 0;
        int col;
        while (s_turn < (colLength - 1) / 2 + rowLength)
        {                               
            col = s_turn - threadIdx.x; 
            if (col >= 0 && col < rowLength)
            {
                if (threadIdx.x == 0)
                {
                    myNum1 = preNum1[0] + fabs(myColNum1 - rowData[col]);

                    if (col == 0)
                    { 
                        myNum2 = myNum1 + fabs(myColNum2 - rowData[col]);
                    }
                    else
                    {
                        myNum2 = min(min(myNum1, preNum1[0]), preNum2[0]) +
                                 fabs(myColNum2 - rowData[col]);
                    }
                }
                else
                {
                    if (col == 0)
                    {
                        myNum1 = preNum2[threadIdx.x - 1] + fabs(myColNum1 - rowData[col]);
                        myNum2 = myNum1 + fabs(myColNum2 - rowData[col]);
                    }
                    else
                    {
                        myNum1 = min(min(prepreNum2[threadIdx.x - 1], preNum2[threadIdx.x - 1]),
                                     preNum1[threadIdx.x]) +
                                 fabs(myColNum1 - rowData[col]);
                        myNum2 = min(min(myNum1, preNum1[threadIdx.x]), preNum2[threadIdx.x]) +
                                 fabs(myColNum2 - rowData[col]);
                    }
                }
            }
            __syncthreads();
            prepreNum2[threadIdx.x] = preNum2[threadIdx.x];
            preNum2[threadIdx.x] = myNum2;
            preNum1[threadIdx.x] = myNum1;
            if (threadIdx.x == 0)
            {
                // printf("--\nI am first thread of %d block,myIdx=%d,turn=%d,col=%d\n",
                // blockIdx.x,
                //      threadIdx.x, s_turn, col);
                s_turn++;
            }
            __syncthreads();
        }
    }
    if (threadIdx.x == (colLength - 1) / 2)
    {
        // if (myNum1 < 50000)
        //    printf("my blockIdx=%d,my result=%f\n", blockIdx.x,
        //    preNum[threadIdx.x]);
        if (colLength % 2 == 0)
            g_odata[blockIdx.y * gridDim.x + blockIdx.x] = myNum2;
        else
            g_odata[blockIdx.y * gridDim.x + blockIdx.x] = myNum1;
    }
}

// vector<vector<float>> GetAlignSignalList(vector<int> &SignalIdList, const int &SignalScale)
// {
//     vector<vector<float>> CenterSignalList;
//     for (int i = 2; i < SignalIdList.size(); i++)
//     {
//         vector<float> TempSignal = GetSingleSignalData(SignalIdList[i], SignalScale);
//         CenterSignalList.push_back(TempSignal);
//     }
//     return CenterSignalList;
// }