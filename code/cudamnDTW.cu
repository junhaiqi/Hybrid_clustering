#include "hip/hip_runtime.h"
﻿//#include "cuda_def.cuh"
//#include "cuda_proc.h"
#include "cuda_def.cuh"
#include <iostream>
#include <vector>
//#include "cuda_kernels.h"
#include <omp.h>
#include <fstream>
#include <string>
#include <stdlib.h>
#include <sstream>
#include <cmath>
#include <numeric>
#include <iomanip>
#include <algorithm>
using namespace std;
#define DBL_MAX 1.7976931348623158e+308 // max value
#define PI 3.1415926535898
#define MAX_NUM 2147483647
#define MAXLENGTH 1023
#define CPUTHREADS 5

using namespace std;
//////////////////////////////////
//函数声明
//////////////////////////////////

__global__ void cuDTW_2048(float *g_allColData, unsigned int *g_allColLength, float *g_allRowData,
                           unsigned int *g_allRowLength, float *g_odata);

float gpuCalculatemnDynamicTimeWarping_2048(const vector<vector<float>> &siga,
                                            const vector<vector<float>> &sigb,
                                            vector<vector<float>> &gpudistResult);
__global__ void cuDTW_ultimate(float *g_allColData,
                               unsigned int *g_allColLength,
                               float *g_allRowData,
                               unsigned int *g_allRowLength, float *g_odata);
void TestDataGeneration(vector<vector<float>> &siga,
                        vector<vector<float>> &sigb, int siga_n = 1000,
                        int sigb_n = 50);
float gpuCalculate1nDynamicTimeWarping(const vector<vector<float>> &siga,
                                       const vector<vector<float>> &sigb,
                                       vector<vector<float>> &gpudistResult);
float cpuCalculate1nDynamicTimeWarping(const vector<vector<float>> &siga,
                                       const vector<vector<float>> &sigb,
                                       vector<vector<float>> &cpudistResult);
float cpuDynamicTimeWarping(const std::vector<float> &seq1,
                            const std::vector<float> &seq2);

// using namespace std;

vector<double> GetSingleSignalData(int &SignalId, const string &argv_1);

template <class T>
void convertFromString(T &value, string &s);

vector<double> GetSingleSignalData1(int &SignalId, const string &argv_2);

vector<int> GetAlignList();

vector<string> GetargvList();

vector<vector<double>> GetCenterSignalList(const vector<int> &AlignList, const string &argv_2);

vector<vector<double>> GetAlignSignalList(vector<int> &SignalIdList, const string &argv_1);

vector<vector<float>> FromDoubleToFloat(vector<vector<double>> a);

void GetminAndpos(vector<vector<float>> &dislist, vector<vector<float>> MinAndPosresultList);

void ZScoreNormalize(vector<double> &signals);

int main()
{
  // int id = 10;
  // int scale = 6126;
  // vector<double > test = GetSingleSignalData1(id,scale);
  // for(int i = 0; i < test.size(); i++){
  //   cout << test[i] << endl;
  // }
  vector<vector<float>> gpudtwresult;
  vector<vector<float>> MinAndPosresultList;
  vector<int> Alignlist = GetAlignList();
  // cout << Alignlist.size() << endl;
  // for(int i = 0; i < Alignlist.size();i++){
  //   cout << Alignlist[i] << endl;
  // }
  // cout << Alignlist[2] << endl;
  int scale = Alignlist[1];
  vector<string> argvList = GetargvList();
  // for(int i = 0; i < test.size(); i++){
  //   cout << test[i] << endl;
  // }
  vector<vector<double>> AlignSignallist = GetAlignSignalList(Alignlist, argvList[0]);
  vector<vector<double>> CenterSignallist = GetCenterSignalList(Alignlist, argvList[1]);
  vector<vector<float>> AlignSignallist1 = FromDoubleToFloat(AlignSignallist);
  vector<vector<float>> CenterSignallist1 = FromDoubleToFloat(CenterSignallist);

  gpuCalculatemnDynamicTimeWarping_2048(AlignSignallist1, CenterSignallist1, gpudtwresult);
  // GetminAndpos(gpudtwresult,MinAndPosresultList);
  // for(int i = 0; i < gpudtwresult.size(); i++){
  //   for(int j = 0; j < gpudtwresult[i].size(); j++){
  //     cout << gpudtwresult[i][j] << endl;
  //   }
  // }
  ofstream disfile("OnetoNdisfile.txt", ios::out);
  string temp;
  for (int i = 0; i < gpudtwresult.size(); i++)
  {
    for (int j = 0; j < gpudtwresult[i].size(); j++)
    {
      temp = std::to_string(gpudtwresult[i][j]);
      disfile << temp;
      disfile << " ";
    }
    disfile << endl;
  }
  // cout << gpudtwresult.size() << endl;
  disfile.close();
  // cout << MinAndPosresultList[0][0] << " " << MinAndPosresultList[1][0] << endl;
  cout << "OnetoNdisfile created successfully!" << endl;
  return 0;
}

//////////////////////////////////
//function
//////////////////////////////////

void ZScoreNormalize(vector<double> &signals)
{
  double sum = accumulate(signals.begin(), signals.end(), 0.0);
  double mean = sum / signals.size();

  double acc = 0.0;
  for (size_t i = signals.size(); i--;)
  {
    signals[i] = signals[i] - mean;
    acc += signals[i] * signals[i];
  }

  double deviation = sqrt(acc / signals.size());

  for (size_t i = signals.size(); i--;)
  {
    signals[i] /= deviation;
  }
}

vector<vector<float>> FromDoubleToFloat(vector<vector<double>> a)
{
  vector<vector<float>> temp;
  for (int i = 0; i < a.size(); i++)
  {
    vector<float> templist;
    for (int j = 0; j < a[i].size(); j++)
    {
      float tempvalue = (float)a[i][j];
      templist.push_back(tempvalue);
    }
    temp.push_back(templist);
  }
  return temp;
}


// void GetminAndpos(const vector<vector<float > >& dislist,vector<vector<float > > MinAndPosresultList){
//    vector<float> minvaluelist;
//    vector<float> positionlist;
//    omp_set_num_threads(CPUTHREADS);
//    #pragma omp parallel for
//    for(int i = 0; i < dislist.size(); i++){
//       float minvalue = dislist[i][0];
//       float position = 0;
//       for(int j = 0; j < dislist[i].size(); j++){
//          if(minvalue > dislist[i][j]){
//            minvalue = dislist[i][j];
//            position = (float)j;
//          }
//       minvaluelist.push_back(minvalue);
//       positionlist.push_back(position);
//       }
//     MinAndPosresultList.push_back(minvaluelist);
//     MinAndPosresultList.push_back(positionlist);
//    }
// }

void GetminAndpos(vector<vector<float>> &dislist, vector<vector<float>> MinAndPosresultList)
{
  vector<float> minvaluelist;
  vector<float> positionlist;
  omp_set_num_threads(CPUTHREADS);
#pragma omp parallel for
  for (int i = 0; i < dislist.size(); i++)
  {
    // float minvalue = 0;
    // float position = 0;
    vector<float>::iterator minvalue = min_element(dislist[i].begin(), dislist[i].end());
    cout << (int)*minvalue << endl;
    minvaluelist.push_back((int)*minvalue);
    positionlist.push_back(distance(dislist[i].begin(), minvalue));
  }
  MinAndPosresultList.push_back(minvaluelist);
  MinAndPosresultList.push_back(positionlist);
}

vector<double> GetSingleSignalData(int &SignalId, const string &argv_1)
{
  // cout << SignalId << endl;
  ifstream SignalFile;
  string TempId = std::to_string(SignalId);
  // string TempId2 = std::to_string(SignalScale);
  string TempString = argv_1 + "/" + "signal_" + TempId + ".txt";
  const char *FileName = TempString.data();
  SignalFile.open(FileName, ios::in);
  if (!SignalFile.is_open())
  {
    cout << "Signal file open error!" << endl;
    cout << argv_1 << endl;
    cout << SignalId << endl;
  }
  string FileLine;
  vector<double> SignalData;
  while (getline(SignalFile, FileLine))
  {
    double SignalValue;
    convertFromString(SignalValue, FileLine);
    SignalData.push_back(SignalValue);
  }
  ZScoreNormalize(SignalData);
  SignalFile.close();
  return SignalData;
}


vector<double> GetSingleSignalData1(int &SignalId, const string &agrv_2)
{
  ifstream SignalFile;
  string TempId = std::to_string(SignalId);
  // string TempId2 = std::to_string(SignalScale);
  string TempString = agrv_2 + "/" + "consensus_sig_" + TempId + ".txt";
  const char *FileName = TempString.data();
  SignalFile.open(FileName, ios::in);
  if (!SignalFile.is_open())
  {
    cout << "CenterSignal file open error!" << endl;
  }
  string FileLine;
  vector<double> SignalData;
  while (getline(SignalFile, FileLine))
  {
    double SignalValue;
    convertFromString(SignalValue, FileLine);
    SignalData.push_back(SignalValue);
  }
  ZScoreNormalize(SignalData);
  SignalFile.close();
  return SignalData;
}

vector<int> GetAlignList()
{
  ifstream Alignfile;
  // const char* filename = "ReadyToSort.txt".data();
  Alignfile.open("ReadyToSortfile.txt", ios::in);
  if (!Alignfile.is_open())
  {
    cout << "Align file open error!" << endl;
  }
  string FileLine;
  vector<int> AlignList;
  while (getline(Alignfile, FileLine))
  {
    int SignalId;
    convertFromString(SignalId, FileLine);
    AlignList.push_back(SignalId);
  }
  return AlignList;
}

vector<string> GetargvList()
{
  ifstream argvfile;
  // const char* filename = "ReadyToSort.txt".data();
  argvfile.open("argv_file.txt", ios::in);
  if (!argvfile.is_open())
  {
    cout << "argv_file.txt open error!" << endl;
  }
  string FileLine;
  vector<string> argvList;
  while (getline(argvfile, FileLine))
  {
    argvList.push_back(FileLine);
  }
  return argvList;
}

vector<vector<double>> GetCenterSignalList(const vector<int> &AlignList, const string &argv_2)
{
  vector<vector<double>> SignalList;
  for (int i = 0; i < AlignList[0]; i++)
  {
    vector<double> TempSignal = GetSingleSignalData1(i, argv_2);
    SignalList.push_back(TempSignal);
  }
  return SignalList;
}

vector<vector<double>> GetAlignSignalList(vector<int> &SignalIdList, const string &argv_1)
{
  vector<vector<double>> CenterSignalList;
  for (int i = 2; i < SignalIdList.size(); i++)
  {
    vector<double> TempSignal = GetSingleSignalData(SignalIdList[i], argv_1);
    CenterSignalList.push_back(TempSignal);
  }
  return CenterSignalList;
}

void TestDataGeneration(vector<vector<float>> &siga,
                        vector<vector<float>> &sigb, int siga_n, int sigb_n)
{
  siga.resize(siga_n);
  sigb.resize(sigb_n);

  omp_set_num_threads(CPUTHREADS);
#pragma omp parallel for
  for (int i = 0; i < siga_n; i++)
  {
    int siga_length = rand() % 200 + 700; // siga_i的长度在700-900之间
    siga[i].resize(siga_length);
    for (int j = 0; j < siga_length; j++)
    {
      siga[i][j] = rand() % 400 + 400; // siga_i_j的范围在400-800之间
    }
  }

#pragma omp parallel for
  for (int i = 0; i < sigb_n; i++)
  {
    int sigb_length = rand() % 200 + 700; // sigb_i的长度在700-900之间
    sigb[i].resize(sigb_length);
    for (int j = 0; j < sigb_length; j++)
    {
      sigb[i][j] = rand() % 400 + 400; // sigb_i_j的范围在400-800之间
    }
  }
}

float gpuCalculate1nDynamicTimeWarping(const vector<vector<float>> &siga,
                                       const vector<vector<float>> &sigb,
                                       vector<vector<float>> &gpudistResult)
{
  // return 0;
  int siga_n = siga.size();
  int sigb_n = sigb.size();
  gpudistResult.resize(siga_n);
  for (int i = 0; i < siga_n; i++)
  {
    gpudistResult[i].resize(sigb_n);
  }
  // int siga_length = 0;
  float *d_distResult = NULL;
  float *d_allColData = NULL;
  float *d_allRowData = NULL;
  unsigned int *d_allRowLength;
  unsigned int *d_allColLength;
  // vector<float *> rowDataList(sigb_n);
  hipMalloc((void **)&d_distResult, siga_n * sigb_n * sizeof(float));
  CUERR
  hipMalloc((void **)&d_allColData, siga_n * 1024 * sizeof(float));
  CUERR
  hipMalloc((void **)&d_allRowData, sigb_n * 1024 * sizeof(float));
  CUERR

  vector<unsigned int> h_allRowLength(sigb_n);
  for (int i = 0; i < sigb_n; i++)
  {
    h_allRowLength[i] = sigb[i].size();
    hipMemcpy(&d_allRowData[1024 * i], &sigb[i][0],
               h_allRowLength[i] * sizeof(float), hipMemcpyHostToDevice);
    CUERR
  }
  hipMalloc((void **)&d_allRowLength, sigb_n * sizeof(unsigned int));
  CUERR
  hipMemcpy(d_allRowLength, &h_allRowLength[0], sigb_n * sizeof(unsigned int),
             hipMemcpyHostToDevice);
  CUERR

  vector<unsigned int> h_allColLength(siga_n);
  for (int i = 0; i < siga_n; i++)
  {
    h_allColLength[i] = siga[i].size();
    hipMemcpy(&d_allColData[1024 * i], &siga[i][0],
               h_allColLength[i] * sizeof(float), hipMemcpyHostToDevice);
    CUERR
  }
  hipMalloc((void **)&d_allColLength, siga_n * sizeof(unsigned int));
  CUERR
  hipMemcpy(d_allColLength, &h_allColLength[0], siga_n * sizeof(unsigned int),
             hipMemcpyHostToDevice);
  CUERR

  float timesum = 0;
  dim3 threadsPerBlock(1024);
  dim3 blocksPerGrid(sigb_n, siga_n);
  cuDTW_ultimate<<<blocksPerGrid, threadsPerBlock>>>(
      d_allColData, d_allColLength, d_allRowData, d_allRowLength, d_distResult);
  CUERR

  for (int i = 0; i < siga_n; i++)
  {
    hipMemcpy(&gpudistResult[i][0], &d_distResult[sigb_n * i],
               sigb_n * sizeof(float), hipMemcpyDeviceToHost);
    CUERR
  }

  hipFree(d_allColData);
  CUERR
  hipFree(d_distResult);
  CUERR
  hipFree(d_allRowData);
  CUERR
  hipFree(d_allRowLength);
  CUERR

  return timesum;
}

float cpuCalculate1nDynamicTimeWarping(const vector<vector<float>> &siga,
                                       const vector<vector<float>> &sigb,
                                       vector<vector<float>> &cpudistResult)
{

  int siga_n = siga.size();
  int sigb_n = sigb.size();
  cpudistResult.resize(siga_n);
  for (int i = 0; i < siga_n; i++)
  {
    cpudistResult[i].resize(sigb_n);
  }
  omp_set_num_threads(CPUTHREADS);
#pragma omp parallel for
  for (int i = 0; i < siga_n; i++)
  {
    for (int j = 0; j < sigb_n; j++)
    {
      cpudistResult[i][j] = cpuDynamicTimeWarping(siga[i], sigb[j]);
    }
  }

  float timesum = 0;
  // printf("cpu Average time use of DTW= %f sec\n", timesum / sigb_n);
  return timesum;
}

float cpuDynamicTimeWarping(const std::vector<float> &seq1,
                            const std::vector<float> &seq2)
{
  vector<vector<float>> score(seq1.size());

  for (int i = 0; i < seq1.size(); i++)
  {
    score[i].resize(seq2.size());
  }

  for (int i = 0; i < seq1.size(); i++)
  {
    for (int j = 0; j < seq2.size(); j++)
    {
      score[i][j] = std::fabs(seq1[i] - seq2[j]);
    }
  }

  for (int i = 1; i < seq1.size(); i++)
  {
    score[i][0] += score[i - 1][0];
  }

  for (int j = 1; j < seq2.size(); j++)
  {
    score[0][j] += score[0][j - 1];
  }

  for (int i = 1; i < seq1.size(); i++)
  {
    for (int j = 1; j < seq2.size(); j++)
    {
      score[i][j] += std::min(std::min(score[i - 1][j], score[i][j - 1]),
                              score[i - 1][j - 1]);
    }
  }

  float diff = score[seq1.size() - 1][seq2.size() - 1];

  return diff;
}

template <class T>
void convertFromString(T &value, string &s)
{
  std::stringstream ss(s);
  ss >> value;
}

__global__ void cuDTW_ultimate(float *g_allColData,
                               unsigned int *g_allColLength,
                               float *g_allRowData,
                               unsigned int *g_allRowLength, float *g_odata)
{

  unsigned int inblockThreadIdx = threadIdx.x;
  unsigned int rowLength =
      g_allRowLength[blockIdx.x]; 
  unsigned int colLength = g_allColLength[blockIdx.y];
  float myNum = 0, myColNum;
  __shared__ unsigned int s_turn;
  __shared__ float preNum[1024], prepreNum[1024], rowData[1024];
  rowData[inblockThreadIdx] =
      g_allRowData[blockIdx.x * blockDim.x + threadIdx.x];
  if (inblockThreadIdx == 0)
  {
    s_turn = 0;
  }
  __syncthreads();
  if (inblockThreadIdx < colLength)
  {
    myColNum = g_allColData[blockIdx.y * 1024 + inblockThreadIdx];
    prepreNum[inblockThreadIdx] = preNum[inblockThreadIdx] = 0;
    int col;
    while (s_turn < colLength + rowLength)
    {
      col = s_turn - inblockThreadIdx;
      if (col >= 0 && col < rowLength)
      {
        if (inblockThreadIdx == 0)
        {
          myNum = preNum[inblockThreadIdx] + fabs(myColNum - rowData[col]);
        }
        else
        {
          if (col == 0)
          {
            myNum =
                preNum[inblockThreadIdx - 1] + fabs(myColNum - rowData[col]);
          }
          else
          {
            myNum = min(min(prepreNum[inblockThreadIdx - 1],
                            preNum[inblockThreadIdx - 1]),
                        preNum[inblockThreadIdx]) +
                    fabs(myColNum - rowData[col]);
          }
        }
      }
      __syncthreads();
      prepreNum[inblockThreadIdx] = preNum[inblockThreadIdx];
      preNum[inblockThreadIdx] = myNum;
      if (inblockThreadIdx == 0)
      {
        // printf("--\nI am first thread of %d block,myIdx=%d,turn=%d,col=%d\n",
        // blockIdx.x,
        //      inblockThreadIdx, s_turn, col);
        s_turn++;
      }
      __syncthreads();
    }
  }
  if (inblockThreadIdx == colLength - 1)
  {
    // if (myNum < 50000)
    //    printf("my blockIdx=%d,my result=%f\n", blockIdx.x,
    //    preNum[inblockThreadIdx]);
    g_odata[blockIdx.y * gridDim.x + blockIdx.x] = myNum;
  }
}

float gpuCalculatemnDynamicTimeWarping_2048(const vector<vector<float>> &siga,
                                            const vector<vector<float>> &sigb,
                                            vector<vector<float>> &gpudistResult)
{
  // return 0;
  int siga_n = siga.size();
  int sigb_n = sigb.size();
  gpudistResult.resize(siga_n);
  for (int i = 0; i < siga_n; i++)
  {
    gpudistResult[i].resize(sigb_n);
  }
  // int siga_length = 0;
  float *d_distResult = NULL;
  float *d_allColData = NULL;
  float *d_allRowData = NULL;
  unsigned int *d_allRowLength;
  unsigned int *d_allColLength;
  // vector<float *> rowDataList(sigb_n);
  hipMalloc((void **)&d_distResult, siga_n * sigb_n * sizeof(float));
  CUERR
  hipMalloc((void **)&d_allColData, siga_n * 2048 * sizeof(float));
  CUERR
  hipMalloc((void **)&d_allRowData, sigb_n * 2048 * sizeof(float));
  CUERR

  
  vector<unsigned int> h_allRowLength(sigb_n);
  for (int i = 0; i < sigb_n; i++)
  {
    h_allRowLength[i] = min(int(sigb[i].size()), 2048);
    hipMemcpy(&d_allRowData[2048 * i], &sigb[i][0], h_allRowLength[i] * sizeof(float),
               hipMemcpyHostToDevice);
    CUERR
  }
  hipMalloc((void **)&d_allRowLength, sigb_n * sizeof(unsigned int));
  CUERR
  hipMemcpy(d_allRowLength, &h_allRowLength[0], sigb_n * sizeof(unsigned int),
             hipMemcpyHostToDevice);
  CUERR


  vector<unsigned int> h_allColLength(siga_n);
  for (int i = 0; i < siga_n; i++)
  {
    h_allColLength[i] = min(int(siga[i].size()), 2048);
    hipMemcpy(&d_allColData[2048 * i], &siga[i][0], h_allColLength[i] * sizeof(float),
               hipMemcpyHostToDevice);
    CUERR
  }
  hipMalloc((void **)&d_allColLength, siga_n * sizeof(unsigned int));
  CUERR
  hipMemcpy(d_allColLength, &h_allColLength[0], siga_n * sizeof(unsigned int),
             hipMemcpyHostToDevice);
  CUERR

  float timesum = 0;
  dim3 threadsPerBlock(1024);
  dim3 blocksPerGrid(sigb_n, siga_n); 
  cuDTW_2048<<<blocksPerGrid, threadsPerBlock>>>(d_allColData, d_allColLength, d_allRowData,
                                                 d_allRowLength, d_distResult);
  CUERR

  for (int i = 0; i < siga_n; i++)
  {
    hipMemcpy(&gpudistResult[i][0], &d_distResult[sigb_n * i], sigb_n * sizeof(float),
               hipMemcpyDeviceToHost);
    CUERR
  }

  hipFree(d_allColData);
  CUERR
  hipFree(d_distResult);
  CUERR
  hipFree(d_allRowData);
  CUERR
  hipFree(d_allRowLength);
  CUERR

  return timesum;
}

__global__ void cuDTW_2048(float *g_allColData, unsigned int *g_allColLength, float *g_allRowData,
                           unsigned int *g_allRowLength, float *g_odata)
{

  unsigned int rowLength = g_allRowLength[blockIdx.x]; 
  unsigned int colLength = g_allColLength[blockIdx.y];
  float myNum1 = 0, myNum2 = 0, myColNum1, myColNum2;
  __shared__ unsigned int s_turn;
  __shared__ float preNum1[1024], preNum2[1024], prepreNum2[1024], rowData[2048];
  
  rowData[threadIdx.x] = g_allRowData[blockIdx.x * 2048 + threadIdx.x];
  __syncthreads();
  rowData[threadIdx.x + 1024] = g_allRowData[blockIdx.x * 2048 + threadIdx.x + 1024];
  if (threadIdx.x == 0)
  {
    s_turn = 0;
  }
  __syncthreads();
  if (threadIdx.x < (colLength - 1) / 2 + 1)
  {
    
    myColNum1 = g_allColData[blockIdx.y * 2048 + (threadIdx.x) * 2];
    myColNum2 = g_allColData[blockIdx.y * 2048 + (threadIdx.x) * 2 + 1];
    prepreNum2[threadIdx.x] = preNum2[threadIdx.x] = preNum1[threadIdx.x] = 0; //初始化
    int col;
    while (s_turn < (colLength - 1) / 2 + 1 + rowLength)
    {                             
      col = s_turn - threadIdx.x; 
      if (col >= 0 && col < rowLength)
      {
        if (threadIdx.x == 0)
        {
          myNum1 = preNum1[0] + fabs(myColNum1 - rowData[col]);

          if (col == 0)
          { 
            myNum2 = myNum1 + fabs(myColNum2 - rowData[col]);
          }
          else
          {
            myNum2 = min(min(myNum1, preNum1[0]), preNum2[0]) +
                     fabs(myColNum2 - rowData[col]);
          }
        }
        else
        {
          if (col == 0)
          {
            myNum1 = preNum2[threadIdx.x - 1] + fabs(myColNum1 - rowData[col]);
            myNum2 = myNum1 + fabs(myColNum2 - rowData[col]);
          }
          else
          {
            myNum1 = min(min(prepreNum2[threadIdx.x - 1], preNum2[threadIdx.x - 1]),
                         preNum1[threadIdx.x]) +
                     fabs(myColNum1 - rowData[col]);
            myNum2 = min(min(myNum1, preNum1[threadIdx.x]), preNum2[threadIdx.x]) +
                     fabs(myColNum2 - rowData[col]);
          }
        }
      }
      __syncthreads();
      prepreNum2[threadIdx.x] = preNum2[threadIdx.x];
      preNum2[threadIdx.x] = myNum2;
      preNum1[threadIdx.x] = myNum1;
      if (threadIdx.x == 0)
      {
        // printf("--\nI am first thread of %d block,myIdx=%d,turn=%d,col=%d\n",
        // blockIdx.x,
        //      threadIdx.x, s_turn, col);
        s_turn++;
      }
      __syncthreads();
    }
  }
  if (threadIdx.x == (colLength - 1) / 2)
  {
    // if (myNum1 < 50000)
    //    printf("my blockIdx=%d,my result=%f\n", blockIdx.x,
    //    preNum[threadIdx.x]);
    if (colLength % 2 == 0)
      g_odata[blockIdx.y * gridDim.x + blockIdx.x] = myNum2;
    else
      g_odata[blockIdx.y * gridDim.x + blockIdx.x] = myNum1;
  }
}